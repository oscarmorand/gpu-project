#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <thread>
#include <cstdio>
#include "logo.h"

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        // We don't exit when we encounter CUDA errors in this example.
        std::exit(EXIT_FAILURE);
    }
}

struct rgb {
    uint8_t r, g, b;
};

struct Lab {
    float L, a, b;
};

int bg_model_pitch;
std::byte* bg_model = nullptr;
int n_images = 0;

__constant__ uint8_t* logo;

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer 
/// @param width 
/// @param height 
/// @param stride 
/// @param pixel_stride 
/// @return 
__global__ void remove_red_channel_inp(std::byte* buffer, int width, int height, int stride)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return; 

    rgb* lineptr = (rgb*) (buffer + y * stride);
    if (y < logo_height && x < logo_width) {
        float alpha = logo[y * logo_width + x] / 255.f;
        lineptr[x].r = 0;
        lineptr[x].g = uint8_t(alpha * lineptr[x].g + (1-alpha) * 255);
        lineptr[x].b = uint8_t(alpha * lineptr[x].b + (1-alpha) * 255);
    } else {
        lineptr[x].r = 0;
    }
}

namespace 
{
    void load_logo()
    {
        static auto buffer = std::unique_ptr<std::byte, decltype(&hipFree)>{nullptr, &hipFree}; 

        if (buffer == nullptr)
        {
            hipError_t err;
            std::byte* ptr;
            err = hipMalloc(&ptr, logo_width * logo_height);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpy(ptr, logo_data, logo_width * logo_height, hipMemcpyHostToDevice);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
            CHECK_CUDA_ERROR(err);

            buffer.reset(ptr);
        }

    }
}

/*
extern "C" {
    void filter_impl(uint8_t* src_buffer, int width, int height, int src_stride, int pixel_stride)
    {
        load_logo();

        assert(sizeof(rgb) == pixel_stride);
        std::byte* dBuffer;
        size_t pitch;

        hipError_t err;
        
        err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        dim3 blockSize(16,16);
        dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);

        remove_red_channel_inp<<<gridSize, blockSize>>>(dBuffer, width, height, pitch);

        err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        hipFree(dBuffer);

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);


        {
            using namespace std::chrono_literals;
            //std::this_thread::sleep_for(100ms);
        }
    }   
}
*/

__device__ Lab rgb_to_lab(rgb in) 
{
    float x = 0.4124564 * in.r + 0.3575761 * in.g + 0.1804375 * in.b;
    float y = 0.2126729 * in.r + 0.7151522 * in.g + 0.0721750 * in.b;
    float z = 0.0193339 * in.r + 0.1191920 * in.g + 0.9503041 * in.b;

    float xn = 95.0489;
    float yn = 100.0;
    float zn = 108.8840;

    float f_y_over_yn = f(y / yn);

    float L = 116 * f_y_over_yn - 16;
    float a = 500 * (f(x / xn) - f_y_over_yn);
    float b = 200 * (f_y_over_yn - f(z / zn));

    return {L, a, b};
}

__global__ void convert_to_lab(std::byte* buffer, std::byte* bg_mask, int width, int height, int src_stride, int bg_mask_pitch)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb* buffer_lineptr = (rgb*) (buffer + y * src_stride);

    Lab lab = rgb_to_lab(buffer_lineptr[x]);

    Lab* bg_mask_lineptr = (Lab*) (bg_mask + y * bg_mask_pitch);
    bg_mask_lineptr[x] = lab;
}

__global__ void handle_first_frame(std::byte* buffer, int width, int height, int stride)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb* lineptr = (rgb*) (buffer + y * stride);

    lineptr[x] = {0,0,0};
}

__global__ void compute_residual_image(std::byte* bg_mask, std::byte* residual_img, int width, int height, int bg_mask_pitch, int residual_img_pitch)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    Lab* bg_mask_lineptr = (Lab*) (bg_mask + y * bg_mask_pitch);
    Lab bg_mask_pixel = bg_mask_lineptr[x];

    Lab* bg_model_lineptr = (Lab*) (bg_model + y * bg_model_pitch);
    Lab bg_model_pixel = bg_model_lineptr[x];

    float* residual_img_lineptr = (float*) (residual_img + y * residual_img_pitch);
    residual_img_lineptr[x] = sqrt(pow(bg_mask_pixel.L - bg_model_pixel.L, 2.0) + pow(bg_mask_pixel.a - bg_model_pixel.a, 2.0) + pow(bg_mask_pixel.b - bg_model_pixel.b, 2.0));
}

__global__ void update_background_model(std::byte* bg_mask, int width, int height, int bg_mask_pitch, int n_frames)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    Lab* bg_mask_lineptr = (Lab*) (bg_mask + y * bg_mask_pitch);
    Lab bg_mask_pixel = bg_mask_lineptr[x];

    Lab* bg_model_lineptr = (Lab*) (bg_model + y * bg_model_pitch);
    Lab bg_model_pixel = bg_model_lineptr[x];

    bg_model_pixel.L = (bg_model_pixel.L * n_frames + bg_mask_pixel.L) / (n_frames + 1);
    bg_model_pixel.a = (bg_model_pixel.a * n_frames + bg_mask_pixel.a) / (n_frames + 1);
    bg_model_pixel.b = (bg_model_pixel.b * n_frames + bg_mask_pixel.b) / (n_frames + 1);

    bg_model_lineptr[x] = bg_model_pixel;
}

#define KERNEL_SIZE 5
    bool kernel[KERNEL_SIZE][KERNEL_SIZE] = {
            {0,1,1,1,0},
            {1,1,1,1,1},
            {1,1,1,1,1},
            {1,1,1,1,1},
            {0,1,1,1,0}};

enum morph_op
{
    EROSION,
    DILATION
};

__global__ void filter_morph(morph_op action, std::byte* residual_img, std::byte* eroded_img, int width, int height, int residual_img_pitch, int eroded_img_pitch)
{
    // TODO Stencil pattern
}

void hysteresis(std::byte* opened_img, std::byte* hyst, int width, int height, int opened_img_pitch, int hyst_pitch)
{
    // TODO Map + Stencil pattern (call two kernels)
    dim3 blockSize(16,16);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);
}

__global__ void masking_output(std::byte* src_buffer, std::byte* hyst, int width, int height, int src_stride, int hyst_pitch)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return;

    rgb* buffer_lineptr = (rgb*) (src_buffer + y * src_stride);
    rgb in_val = buffer_lineptr[x];

    bool* hyst_lineptr = (bool*) (hyst + y * hyst_pitch);
    bool val = hyst_lineptr[x];

    buffer_lineptr[x].r = in_val.r / 2 + (val ? 127 : 0);
    buffer_lineptr[x].g = in_val.g / 2;
    buffer_lineptr[x].b = in_val.b / 2;
}

extern "C" {
    void filter_impl(uint8_t* src_buffer, int width, int height, int src_stride, int pixel_stride)
    {
        assert(sizeof(rgb) == pixel_stride);
        hipError_t err;

        dim3 blockSize(16,16);
        dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);
        
        size_t bg_mask_pitch;
        std::byte* bg_mask;
        err = hipMallocPitch(&bg_mask, &bg_mask_pitch, width * sizeof(Lab), height);
        CHECK_CUDA_ERROR(err);

        // Conversion from RGB to Lab color space
        convert_to_lab<<<gridSize, blockSize>>>(src_buffer, bg_mask, width, height, src_stride, bg_mask_pitch);
        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);

        if (bg_model == nullptr) // First frame, no background model
        {
            err = hipMallocPitch(&bg_model, &bg_model_pitch, width * sizeof(Lab), height);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpy2D(bg_model, bg_model_pitch, bg_mask, bg_mask_pitch, width * sizeof(Lab), height, hipMemcpyDeviceToDevice);
            CHECK_CUDA_ERROR(err);

            handle_first_frame<<<gridSize, blockSize>>>(src_buffer, width, height, src_stride);
            err = hipDeviceSynchronize();
            CHECK_CUDA_ERROR(err);
        }
        else // Normal case
        {
            size_t residual_img_pitch;
            std::byte* residual_img;
            err = hipMallocPitch(&residual_img, &residual_img_pitch, width * sizeof(Lab), height);
            CHECK_CUDA_ERROR(err);

            // Compute the residual image
            compute_residual_image<<<gridSize, blockSize>>>(bg_mask, residual_img, width, height, bg_mask_pitch, residual_img_pitch);
            err = hipDeviceSynchronize();
            CHECK_CUDA_ERROR(err);

            // Update the background model with the computed background mask
            update_background_model<<<gridSize, blockSize>>>(bg_mask, width, height, bg_mask_pitch, n_images);
            err = hipDeviceSynchronize();
            CHECK_CUDA_ERROR(err);
            n_images += 1;

            // Erosion
            size_t eroded_img_pitch;
            std::byte* eroded_img;
            err = hipMallocPitch(&eroded_img, &eroded_img_pitch, width * sizeof(float), height);
            CHECK_CUDA_ERROR(err);
            filter_morph<<<gridSize, blockSize>>>(EROSION, residual_img, eroded_img, width, height, residual_img_pitch, eroded_img_pitch);
            err = hipDeviceSynchronize();
            CHECK_CUDA_ERROR(err);
            hipFree(residual_img);

            // Dilation
            size_t opened_img_pitch;
            std::byte* opened_img;
            err = hipMallocPitch(&opened_img, &opened_img_pitch, width * sizeof(float), height);
            CHECK_CUDA_ERROR(err);
            filter_morph<<<gridSize, blockSize>>>(DILATION, residual_img, opened_img, width, height, residual_img_pitch, opened_img_pitch);
            err = hipDeviceSynchronize();
            CHECK_CUDA_ERROR(err);
            hipFree(eroded_img);

            // Hysteresis
            size_t hyst_pitch;
            std::byte* hyst;
            err = hipMallocPitch(&hyst, &hyst_pitch, width * sizeof(bool), height);
            CHECK_CUDA_ERROR(err);
            hysteresis(opened_img, hyst, width, height, opened_img_pitch, hyst_pitch);
            hipFree(opened_img);

            // Save the mask
            masking_output<<<gridSize, blockSize>>>(src_buffer, hyst, width, height, src_stride, hyst_pitch);
            err = hipDeviceSynchronize();
            CHECK_CUDA_ERROR(err);

            hipFree(hyst);
        }

        hipFree(bg_mask);
    }   
}