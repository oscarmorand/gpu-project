#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <thread>
#include <cstdio>
#include "logo.h"

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        // We don't exit when we encounter CUDA errors in this example.
        std::exit(EXIT_FAILURE);
    }
}

struct rgb {
    uint8_t r, g, b;
};

__constant__ uint8_t* logo;

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer 
/// @param width 
/// @param height 
/// @param stride 
/// @param pixel_stride 
/// @return 
__global__ void remove_red_channel_inp(std::byte* buffer, int width, int height, int stride)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return; 

    rgb* lineptr = (rgb*) (buffer + y * stride);
    if (y < logo_height && x < logo_width) {
        float alpha = logo[y * logo_width + x] / 255.f;
        lineptr[x].r = 0;
        lineptr[x].g = uint8_t(alpha * lineptr[x].g + (1-alpha) * 255);
        lineptr[x].b = uint8_t(alpha * lineptr[x].b + (1-alpha) * 255);
    } else {
        lineptr[x].r = 0;
    }
}




namespace 
{
    void load_logo()
    {
        static auto buffer = std::unique_ptr<std::byte, decltype(&hipFree)>{nullptr, &hipFree}; 

        if (buffer == nullptr)
        {
            hipError_t err;
            std::byte* ptr;
            err = hipMalloc(&ptr, logo_width * logo_height);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpy(ptr, logo_data, logo_width * logo_height, hipMemcpyHostToDevice);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
            CHECK_CUDA_ERROR(err);

            buffer.reset(ptr);
        }

    }
}

extern "C" {
    void filter_impl(uint8_t* src_buffer, int width, int height, int src_stride, int pixel_stride)
    {
        load_logo();

        assert(sizeof(rgb) == pixel_stride);
        std::byte* dBuffer;
        size_t pitch;

        hipError_t err;
        
        err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        dim3 blockSize(16,16);
        dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);

        remove_red_channel_inp<<<gridSize, blockSize>>>(dBuffer, width, height, pitch);

        err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        hipFree(dBuffer);

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);


        {
            using namespace std::chrono_literals;
            //std::this_thread::sleep_for(100ms);
        }
    }   
}
